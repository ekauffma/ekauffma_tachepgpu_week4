
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

const int DSIZE = 256;
const int a = 2;
const int b = 4;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)


// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        atomicAdd(C, A[idx] * B[idx]);
    }

}


int main() {

	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 

	h_A = new int[DSIZE];
	h_B = new int[DSIZE];
	h_C = new int;
	for (int i = 0; i < DSIZE; i++){
		h_A[i] = a;
    	h_B[i] = b;
	}
	*h_C = 0;

	// Allocate device memory 
    hipMalloc(&d_A, DSIZE * sizeof(int));
    hipMalloc(&d_B, DSIZE * sizeof(int));
    hipMalloc(&d_C, sizeof(int));
	
	// Check memory allocation for errors
    cudaCheckErrors();

	// Copy the matrices on GPU
    hipMemcpy(d_A, h_A, DSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, sizeof(int), hipMemcpyHostToDevice);
	
	// Check memory copy for errors
    cudaCheckErrors();

	// Define block/grid dimensions and launch kernel
    int blockSize = 32;
    int gridSize = DSIZE / blockSize;
    dot_product<<<gridSize, blockSize>>>(d_A, d_B, d_C, DSIZE);
	
	// Copy results back to host
    hipMemcpy(h_A, d_A, DSIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, DSIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, sizeof(int), hipMemcpyDeviceToHost);

    // Check copy for errors
    cudaCheckErrors();

	// Verify result
    printf("h_A = [");
    for (int i=0; i < DSIZE; i++) {
        printf("%d, ",h_A[i]);
    }
    printf("]\n\n");
    printf("h_B = [");
    for (int i=0; i < DSIZE; i++) {
        printf("%d, ",h_B[i]);
    }
    printf("]\n\n");
    printf("h_C = %d \n\n", *h_C);

	// Free allocated memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
	
	return 0;

}
